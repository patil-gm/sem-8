
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
using namespace std;

// CUDA code to multiply matrices
__global__ void multiply(int* A, int* B, int* C, int size) {
    // Uses thread indices and block indices to compute each element
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < size && col < size) {
        int sum = 0;
        for (int i = 0; i < size; i++) {
            sum += A[row * size + i] * B[i * size + col];
        }
        C[row * size + col] = sum;
    }
}

void initialize(vector<vector<int>>& matrix, int size) {
    cout << "Enter the elements of the matrix (" << size << "x" << size << "):\n";
    for (int row = 0; row < size; row++) {
        for (int col = 0; col < size; col++) {
            cin >> matrix[row][col];
        }
    }
}

void print(int* matrix, int size) {
    for (int row = 0; row < size; row++) {
        for (int col = 0; col < size; col++) {
            cout << matrix[row * size + col] << " ";
        }
        cout << '\n';
    }
    cout << '\n';
}

int main() {
    int* A, * B, * C;

    int N;
    cout << "Enter the size of the square matrices: ";
    cin >> N;

    vector<vector<int>> A_vec(N, vector<int>(N));
    vector<vector<int>> B_vec(N, vector<int>(N));
    vector<vector<int>> C_vec(N, vector<int>(N));

    initialize(A_vec, N);
    initialize(B_vec, N);
    cout << "Matrix A: \n";
    for (int row = 0; row < N; row++) {
        for (int col = 0; col < N; col++) {
            cout << A_vec[row][col] << " ";
        }
        cout << '\n';
    }
    cout << '\n';

    cout << "Matrix B: \n";
    for (int row = 0; row < N; row++) {
        for (int col = 0; col < N; col++) {
            cout << B_vec[row][col] << " ";
        }
        cout << '\n';
    }
    cout << '\n';

    int matrixSize = N * N;
    size_t matrixBytes = matrixSize * sizeof(int);

    A = new int[matrixSize];
    B = new int[matrixSize];
    C = new int[matrixSize];

    // Copy data from vectors to arrays
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            A[i * N + j] = A_vec[i][j];
            B[i * N + j] = B_vec[i][j];
        }
    }

    int* X, * Y, * Z;
    // Allocate space
    hipMalloc(&X, matrixBytes);
    hipMalloc(&Y, matrixBytes);
    hipMalloc(&Z, matrixBytes);

    // Copy values from A to X
    hipMemcpy(X, A, matrixBytes, hipMemcpyHostToDevice);

    // Copy values from A to X and B to Y
    hipMemcpy(Y, B, matrixBytes, hipMemcpyHostToDevice);

    // Threads per CTA dimension
    int THREADS = 2;

    // Blocks per grid dimension (assumes THREADS divides N evenly)
    int BLOCKS = N / THREADS;

    // Use dim3 structs for block  and grid dimensions
    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);

    // Launch kernel
    multiply<<<blocks, threads>>>(X, Y, Z, N);

    hipMemcpy(C, Z, matrixBytes, hipMemcpyDeviceToHost);
    cout << "Multiplication of matrix A and B: \n";
    print(C, N);

    delete[] A;
    delete[] B;
    delete[] C;

    hipFree(X);
    hipFree(Y);
    hipFree(Z);

    return 0;
}